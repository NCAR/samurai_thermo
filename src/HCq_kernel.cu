
#include <hip/hip_runtime.h>
#include <stdio.h>
#ifndef _CSTGPU_KERNEL_H
#define _CSTGPU_KERNEL_H
#define BLOCKSIZE 6

// Declare some variables
__constant__ float BC[9][4];
__constant__ float coeffDevice[8500];
float* obs_d;
float* HCq_d;

// Basis Functions
__device__ float Basis(int m, float x, int M, float xmin, 
						float DX, float DXrecip, float ONESIXTH, int C)
{
	
	float b = 0;
	float xm = xmin + (m * DX);
	float delta = (x - xm) * DXrecip;
	float z = fabsf(delta);
	
	if (z < 2.0)
	{
		z = 2 - z;
		b = (z*z*z) * ONESIXTH;
		z -= 1.0;
		if (z > 0)
			b -= (z*z*z) * 4 * ONESIXTH;
	}
	
	// Boundary conditions, if any, are an additional addend.
	if (m == 0 || m == 1) {
		float l = 0;
		xm = xmin + (-1 * DX);
		delta = (x - xm) * DXrecip;
		z = fabsf(delta);
		
		if (z < 2.0)
		{
			z = 2 - z;
			l = (z*z*z) * ONESIXTH;
			z -= 1.0;
			if (z > 0)
				l -= (z*z*z) * 4 * ONESIXTH;
		}
		b += BC[C][m] * l;
	} else if (m == M-1 || m == M) {
		float r = 0;
		xm = xmin + ((M+1) * DX);
		delta = (x - xm) * DXrecip;
		z = fabsf(delta);
		
		if (z < 2.0)
		{
			z = 2 - z;
			r = (z*z*z) * ONESIXTH;
			z -= 1.0;
			if (z > 0)
				r -= (z*z*z) * 4 * ONESIXTH;
		}
		b += BC[C][m+3-M] * r;
	}
	return b;
}

__device__ float DBasis(int m, float x, int M, float xmin, 
						float DX, float DXrecip, float ONESIXTH, int C)
{
	float b = 0;
	float xm = xmin + (m * DX);
	float delta = (x - xm) * DXrecip;
	float z = fabsf(delta);
	
	if (z < 2.0)
	{
		z = 2.0 - z;
		b = (z*z) * ONESIXTH;
		z -= 1.0;
		if (z > 0)
			b -= (z*z) * 4 * ONESIXTH;
		b *= ((delta > 0) ? -1.0 : 1.0) * 3.0 / DX;
	}
	
	// Boundary conditions, if any, are an additional addend.
	if (m == 0 || m == 1) {
		float l = 0;
		xm = xmin + (-1 * DX);
		delta = (x - xm) * DXrecip;
		z = fabsf(delta);
		
		if (z < 2.0)
		{
			z = 2 - z;
			l = (z*z) * ONESIXTH;
			z -= 1.0;
			if (z > 0)
				l -= (z*z) * 4 * ONESIXTH;
			l *= ((delta > 0) ? -1.0 : 1.0) * 3.0 / DX;
		}
		
		b += BC[C][m] * l;
	} else if (m == M-1 || m == M) {
		float r = 0;
		xm = xmin + ((M+1) * DX);
		delta = (x - xm) * DXrecip;
		z = fabsf(delta);
		
		if (z < 2.0)
		{
			z = 2 - z;
			r = (z*z) * ONESIXTH;
			z -= 1.0;
			if (z > 0)
				r -= (z*z) * 4 * ONESIXTH;
			r *= ((delta > 0) ? -1.0 : 1.0) * 3.0 / DX;	
		}
		b += BC[C][m+3-M] * r;
	}
	return b;
}

__global__ void HCq_kernel(float* obs_d,float* HCq_d, int R, int Z, float rmin, float DR, float DRrecip, 
						   float zmin, float DZ, float DZrecip, float ONESIXTH)
{

	// Block and thread indices
	//int bx = blockIdx.x;
	//int tx = threadIdx.x;
	int xi = blockIdx.x*BLOCKSIZE + threadIdx.x;
	int mi = xi*9;
	int R1 = R+1;
	float w1 = obs_d[mi];
	float w2 = obs_d[mi+1];
	float w3 = obs_d[mi+2];
	float w4 = obs_d[mi+3];
	float w5 = obs_d[mi+4];
	float w6 = obs_d[mi+5];
	float radius = obs_d[mi+6];
	float height = obs_d[mi+7];
	float invRadius = 1./radius;
	float HCq = 0;
	int m = (int)((radius - rmin)*DRrecip);
	int n = (int)((height - zmin)*DZrecip);
	float bz = 0;
	float br = 0;
	float bzp = 0;
	float brp = 0;
	int bc = 1;
	// rhoV = BC_LZERO_RSECOND, r & BC_ZERO_SECOND, z
	for (int r = m-1; r <= m+2; ++r) {
		for (int z = n-1; z <= n+2; ++z) {				
			if ((r < 0) or (r > R) or (z < 0) or (z > Z)) continue;
			if ((r > 1) and (r < R-1) and (z > 1) and (z < Z-1)) {
				// No BCs to worry about, calculate the basis once
				br = Basis(r, radius, R, rmin, DR, DRrecip, ONESIXTH, 2);
				bz = Basis(z, height, Z, zmin, DZ, DZrecip, ONESIXTH, 2);
				brp = DBasis(r, radius, R, rmin, DR, DRrecip, ONESIXTH, 4);
				bzp = DBasis(z, height, Z, zmin, DZ, DZrecip, ONESIXTH, 4);
				//printf("%d,  %d, %f, %f\n", r, z, br, bz);
				bc = 0;
				HCq += coeffDevice[z*5*R1 + r*5] * br * bz * w1;
				float coeff = coeffDevice[z*5*R1 + r*5 +1];
				HCq += coeff * br * (-bzp) * w2 * 1e3 * invRadius;
				HCq += coeff * brp * bz * w3 * invRadius;
				HCq += coeffDevice[z*5*R1 + r*5 +2] * br * bz * w4;
				HCq += coeffDevice[z*5*R1 + r*5 +3] * br * bz * w5;
				HCq += coeffDevice[z*5*R1 + r*5 +4] * br * bz * w6;
			} else {
				if (w1) {
					if (bc) { 
						br = Basis(r, radius, R, rmin, DR, DRrecip, ONESIXTH, 4);
						bz = Basis(z, height, Z, zmin, DZ, DZrecip, ONESIXTH, 2);
					}
					HCq += coeffDevice[z*5*R1 + r*5] * br * bz * w1;
				}
				float coeff;
				if (w2 or w3) coeff = coeffDevice[z*5*R1 + r*5 +1];
				if (w2) {
					if (bc) {
						br = Basis(r, radius, R, rmin, DR, DRrecip, ONESIXTH, 4);
						bzp = DBasis(z, height, Z, zmin, DZ, DZrecip, ONESIXTH, 4);
					}
					HCq += coeff * br * (-bzp) * w2 * 1e3 * invRadius;
				}
				if (w3) {
					if (bc) {
						brp = DBasis(r, radius, R, rmin, DR, DRrecip, ONESIXTH, 4);
						bz = Basis(z, height, Z, zmin, DZ, DZrecip, ONESIXTH, 4);
					}
					HCq += coeff * brp * bz * w3 * invRadius;
				}
				if (w4 or w5 or w6) {
					if (bc) {
						br = Basis(r, radius, R, rmin, DR, DRrecip, ONESIXTH, 2);
						bz = Basis(z, height, Z, zmin, DZ, DZrecip, ONESIXTH, 2);
					}
				}
				if (w4)
					HCq += coeffDevice[z*5*R1 + r*5 +2] * br * bz * w4;
				if (w5)
					HCq += coeffDevice[z*5*R1 + r*5 +3] * br * bz * w5;
				if (w6)
					HCq += coeffDevice[z*5*R1 + r*5 +4] * br * bz * w6;
			}
		}
	}
	HCq_d[xi] = HCq;
	//printf("%d : %f\n", xi,  HCq);
}

#endif
